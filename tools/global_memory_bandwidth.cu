#include "hip/hip_runtime.h"
#include <stdio.h>

#include "utils/cuda_event.h"
#include "utils/macros.h"

__global__ void Saxpy(int n, float a, float *x, float *y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    y[i] = a * x[i] + y[i];
  }
}

int main() {
  int device_num;

  hipGetDeviceCount(&device_num);
  for (int i = 0; i < device_num; i++) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, i));
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
  }

  const int N = 20 * (1 << 20);
  float *x, *y, *d_x, *d_y;

  x = (float *)malloc(N * sizeof(float));
  y = (float *)malloc(N * sizeof(float));
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
  CUDA_CHECK(hipMalloc(&d_x, N * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_y, N * sizeof(float)));

  CudaEvent start, stop;

  CUDA_CHECK(hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice));

  CUDA_CHECK(hipEventRecord(start.get()));
  Saxpy<<<(N + 511) / 512, 512>>>(N, 2.0f, d_x, d_y);
  CUDA_CHECK(hipEventRecord(stop.get()));

  hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

  hipEventSynchronize(stop.get());

  float elapsed_time_ms = 0;
  hipEventElapsedTime(&elapsed_time_ms, start.get(), stop.get());

  printf("Effective Bandwidth (GB/s): %f\n", N * 4 * 3 / elapsed_time_ms / 1e6);

  free(x);
  free(y);
  hipFree(d_x);
  hipFree(d_y);

  return 0;
}
