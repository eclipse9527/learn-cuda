#include "hip/hip_runtime.h"
#include <stdio.h>

#include "utils/cuda_event.h"
#include "utils/macros.h"

__global__ void Saxpy(int n, float *x, float *y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    y[i] = x[i] + y[i];
  }
}

int main() {
  int device_num;

  hipGetDeviceCount(&device_num);
  for (int i = 0; i < device_num; i++) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, i));
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
  }

  const int N = 20 * (1 << 20);
  float *x, *y, *d_x, *d_y;

  x = (float *)malloc(N * sizeof(float));
  y = (float *)malloc(N * sizeof(float));
  CUDA_CHECK(hipMalloc(&d_x, N * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_y, N * sizeof(float)));

  CudaEvent start, stop;

  CUDA_CHECK(hipEventRecord(start.get()));
  Saxpy<<<(N + 511) / 512, 512>>>(N, d_x, d_y);
  CUDA_CHECK(hipEventRecord(stop.get()));

  CUDA_CHECK(hipEventSynchronize(stop.get()));

  float elapsed_time_ms = 0;
  CUDA_CHECK(hipEventElapsedTime(&elapsed_time_ms, start.get(), stop.get()));

  printf("Effective Bandwidth (GB/s): %f\n", N * 4 * 3 / elapsed_time_ms / 1e6);

  free(x);
  free(y);
  hipFree(d_x);
  hipFree(d_y);

  return 0;
}
